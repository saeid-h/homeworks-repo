#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#define ARRAYSIZE 32

//Note: _global_: from host and run in the device(such function is called kernel),and the result of kernel is stored in GPU and explicitily transferred to host, a function has no directive will run in host(by default)

//1.read in rows, write in columns
__global__ void row_col(int* odata, int* idata, int n) {
	int i;
	for (i = 0; i < n; i++) {
		odata[i*n + blockIdx.x] = idata[blockIdx.x*n + i];
	}
}

//2.read in columns and write in rows
__global__ void col_row(int* odata, int* idata, int n) {
	int i;
	for (i = 0; i < n; i++) {
		odata[blockIdx.x*n + i] = idata[i*n + blockIdx.x];
	}
}
//3.read in rows and write in columns + unroll 4 blocks
__global__ void row_col_unroll(int* odata, int* idata, int n) {
	for (int i = 0; i < 4; i++) {
		int x = blockIdx.x * 4 + i;
		for (int j = 0; j < n; j++) {
			odata[j*n + x] = idata[x*n + j];
		}
	}

}
//4.read in columns and write in rows + unroll 4 blocks
__global__ void col_row_unroll(int* odata, int* idata, int n) {
	for (int i = 0; i < 4; i++) {
		int x = blockIdx.x * 4 + i;
		for (int j = 0; j < n; j++) {
			odata[x*n + j] = idata[j*n + x];
		}
	}
}
//5.read in rows and write in columns + diagonal
__global__ void row_col_diag(float *odata, float *idata, int n) {
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < 4; j++) {
			int x = i;
			int y = (i + blockIdx.x * 4 + j) % n;
			odata[y*n + x] = idata[x*n + y];
		}
	}
}
//6.read in columns and write in row + diagonal
__global__ void col_row_diag(int* odata, int* idata, int n) {
		for (int i = 0; i < n; i++) {
			for (int j = 0; j < 4; j++) {
				int x = i;
				int y = (i + blockIdx.x * 4 + j) % n;
				odata[y*n + x] = idata[x*n + y];
			}
		}
	}
void print_matrix(int* h_tdata, int n) {
	int i, j;
	printf("print matrix of %dx%d\n", n, n);
	for (i = 0; i < n; i++) {
		for (j = 0; j < n; j++) {
			printf("%d ", h_tdata[i*n + j]);
		}
		printf("\n");
	}
}

int main(int argc, char **argv) {
	int m_th, m_size;
	if (argc > 2) {
		m_th = atoi(argv[1]);
		m_size = atoi(argv[2]);
	}
	int i, j;
	const int n = m_size;
	const int mem_size = n*n * sizeof(int);

	//allocate memory for the matrix in host(including input and output)
	int *h_idata = (int*)malloc(mem_size);
	int *h_tdata = (int*)malloc(mem_size);
	//allocate memory for the matrix in device(including input and output)
	int *d_idata, *d_tdata;
	hipMalloc(&d_idata, mem_size);
	hipMalloc(&d_tdata, mem_size);

	//produce the matrix for transposition 
	for (i = 0; i < n; i++) {
		for (j = 0; j < n; j++) {
			h_idata[i*n + j] = i*n + j;
		}
	}

	/*-------------------------preparation------------------------------*/
	hipMemcpy(d_idata, h_idata, mem_size, hipMemcpyHostToDevice);//copy data from host to device(GPU)
																   //events for timing
	hipEvent_t startEvent, stopEvent;
	hipEventCreate(&startEvent);
	hipEventCreate(&stopEvent);
	float ms;
	hipMemset(d_tdata, 0, mem_size);
	
	/*-------------------implement different method-----------------------------*/
	hipEventRecord(startEvent, 0);//start timing
	dim3 dimGrid(n, 1);
	dim3 dimBlock(1, 1);
	//to do 
	switch (m_th) {
		case 1:
			row_col << <dimGrid, 1 >> > (d_tdata, d_idata, n);//<<<grid,block>>> grid: each column is a block
			break;
		case 2:
			col_row << <dimGrid, 1 >> > (d_tdata, d_idata, n);
			break;
		case 3:
			row_col_unroll << < dimGrid, 1 >> > (d_tdata, d_idata, n);
			break;
		case 4:
			col_row_unroll << < dimGrid, 1 >> > (d_tdata, d_idata, n);
			break;
		case 5:
			row_col_diag << < dimGrid, 1 >> > (d_tdata, d_idata, n);
			break;
		case 6:
			col_row_diag << < dimGrid, 1 >> > (d_tdata, d_idata, n);
	}	

	/*--------------------------------------------------------------------------*/
	hipEventRecord(stopEvent, 0);//end timing
	hipEventSynchronize(stopEvent);//stop timing
	hipEventElapsedTime(&ms, startEvent, stopEvent);
	hipMemcpy(h_tdata, d_tdata, mem_size, hipMemcpyDeviceToHost);//copy data from device(GPU) to host

	/*print_matrix(h_idata, n);
	print_matrix(h_tdata, n);*/

	//calculate the elapsed time 
	printf("the elapsed time is:%.10f\n", ms);

	/*------------------ending work:release memory in GPU and heap-----------------*/
	hipEventDestroy(startEvent);
	hipEventDestroy(stopEvent);
	hipFree(d_tdata);
	hipFree(d_idata);
	free(h_idata);
	free(h_tdata);
	return 0;
}
