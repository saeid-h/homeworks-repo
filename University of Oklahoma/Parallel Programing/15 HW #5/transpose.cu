#include "hip/hip_runtime.h"



#include	<stdio.h>
#include	<stdlib.h>
#include	<hip/hip_runtime.h>
#include	<string.h>
#include "hip/hip_runtime.h"
#include ""

#define ARRAYSIZE 32
#define TILE_DIM 	32
#define BLOCK_ROWS 	8
#define NUM_REPS  100

__global__ void helloWorld (float *a, float *b, int size) {
	
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (id < size*size){
		a[id] += b[id];
	}
	
}

__global__ void copy(float *odata, float* idata, int width,
                                     int height) {
										 
	int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
	int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;
	
	int index  = xIndex + width * yIndex;
	
	for (int i = 0; i < TILE_DIM; i += BLOCK_ROWS) {
		odata[index+i*width] = idata[index+i*width];
  	}
}



__global__ void transposeNaive (float *odata, float* idata, 
								int width, int height, char direction) {
  
	int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
	int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;
	
	int index_in  = xIndex + width * yIndex;
	int index_out = yIndex + height * xIndex;
	
	for (int i=0; i < TILE_DIM; i += BLOCK_ROWS) {
		if ( direction == 'H')
			odata[index_out+i] = idata[index_in+i*width];
		else
			odata[index_out+i*width] = idata[index_in+i];
	}
}



__global__ void row_col(int* odata, int* idata, int n) {
	int i;
	for (i = 0; i < n; i++) {
		odata[i*n + blockIdx.x] = idata[blockIdx.x*n + i];
	}
}

__global__ void col_row(int* odata, int* idata, int n) {
	int i;
	for (i = 0; i < n; i++) {
		odata[blockIdx.x*n + i] = idata[i*n + blockIdx.x];
	}
}
__global__ void row_col_unroll(int* odata, int* idata, int n) {
	for (int i = 0; i < 4; i++) {
		int x = blockIdx.x * 4 + i;
		for (int j = 0; j < n; j++) {
			odata[j*n + x] = idata[x*n + j];
		}
	}

}


__global__ void col_row_unroll(int* odata, int* idata, int n) {
	for (int i = 0; i < 4; i++) {
		int x = blockIdx.x * 4 + i;
		for (int j = 0; j < n; j++) {
			odata[x*n + j] = idata[j*n + x];
		}
	}
}

__global__ void row_col_diag(int *odata, int *idata, int n) {
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < 4; j++) {
			int x = i;
			int y = (i + blockIdx.x * 4 + j) % n;
			odata[y*n + x] = idata[x*n + y];
		}
	}
}

__global__ void col_row_diag(int* odata, int* idata, int n) {
		for (int i = 0; i < n; i++) {
			for (int j = 0; j < 4; j++) {
				int x = i;
				int y = (i + blockIdx.x * 4 + j) % n;
				odata[y*n + x] = idata[x*n + y];
			}
		}
	}
	
	
void print_matrix(int* h_tdata, int n) {
	int i, j;
	printf("print matrix of %dx%d\n", n, n);
	for (i = 0; i < n; i++) {
		for (j = 0; j < n; j++) {
			printf("%d ", h_tdata[i*n + j]);
		}
		printf("\n");
	}
}


  
  
__global__ void transposeDiagonal (float *odata,
            float *idata, int width, int height, char direction) {
				
	__shared__ float tile[TILE_DIM][TILE_DIM+1];
	int blockIdx_x, blockIdx_y;
	
	// diagonal reordering
	if (width == height) {
		blockIdx_y = blockIdx.x;
		blockIdx_x = (blockIdx.x+blockIdx.y)%gridDim.x;
	} else {
		int bid = blockIdx.x + gridDim.x*blockIdx.y;
		blockIdx_y = bid%gridDim.y;
		blockIdx_x = ((bid/gridDim.y)+blockIdx_y)%gridDim.x;
	}
	
	int xIndex = blockIdx_x*TILE_DIM + threadIdx.x;
	int yIndex = blockIdx_y*TILE_DIM + threadIdx.y;
	int index_in = xIndex + (yIndex) * width;
	
	xIndex = blockIdx_y*TILE_DIM + threadIdx.x;
	yIndex = blockIdx_x*TILE_DIM + threadIdx.y;
	
	int index_out = xIndex + (yIndex)*height;
	for (int i = 0; i < TILE_DIM; i += BLOCK_ROWS) {
		if (direction == 'H')
			tile[threadIdx.y+i][threadIdx.x] = idata[index_in+i*width];
		else
			tile[threadIdx.y+i][threadIdx.x] = idata[index_in+i*height];
	}
	
	__syncthreads();
	
	for (int i = 0; i < TILE_DIM; i += BLOCK_ROWS) {
		if (direction == 'H')
			odata[index_out+i*height] = tile[threadIdx.x][threadIdx.y+i];
		else
			odata[index_out+i*width] = tile[threadIdx.x][threadIdx.y+i];
	}
}




__global__ void transposeCoalesced (float *odata,
            float *idata, int width, int height) {

	__shared__ float tile[TILE_DIM][TILE_DIM];
	int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
	int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;
	
	int index_in = xIndex + (yIndex) * width;
	
	xIndex = blockIdx.y * TILE_DIM + threadIdx.x;
	yIndex = blockIdx.x * TILE_DIM + threadIdx.y;
	
	int index_out = xIndex + (yIndex) * height;
	
	for (int i = 0; i < TILE_DIM; i += BLOCK_ROWS) {
		tile[threadIdx.y+i][threadIdx.x] = idata[index_in+i*width];
	}
	
	__syncthreads();
	for (int i = 0; i < TILE_DIM; i += BLOCK_ROWS) {
		odata[index_out+i*height] = tile[threadIdx.x][threadIdx.y+i];
	}
}




__global__ void copySharedMem (float *odata, float *idata,
                          int width, int height) {
							  
	__shared__ float tile[TILE_DIM][TILE_DIM];
	int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
	int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;
	int index  = xIndex + width * yIndex;
	
	for (int i = 0; i < TILE_DIM; i += BLOCK_ROWS) {
		tile[threadIdx.y+i][threadIdx.x] = idata[index+i*width];
	}
	
	__syncthreads();
	for (int i=0; i<TILE_DIM; i+=BLOCK_ROWS) {
		odata[index+i*width] = tile[threadIdx.y+i][threadIdx.x];
	}
}



__global__ void transposeFineGrained (float *odata,
           float *idata, int width, int height) {
			   
	__shared__ float block[TILE_DIM][TILE_DIM+1];
	int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
	int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;
	
	int index = xIndex + (yIndex) * width;
	
	for (int i = 0; i < TILE_DIM; i += BLOCK_ROWS) {
		block[threadIdx.y+i][threadIdx.x] = idata[index+i*width];
	}
	
	__syncthreads();
  
	for (int i = 0; i < TILE_DIM; i += BLOCK_ROWS) {
		odata[index+i*height] = block[threadIdx.x][threadIdx.y+i];
    }
}




__global__ void transposeCoarseGrained (float *odata,
        float *idata, int width, int height) {
						
	__shared__ float block[TILE_DIM][TILE_DIM+1];
	int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
    int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;
	
    int index_in = xIndex + (yIndex) * width;
	
    xIndex = blockIdx.y * TILE_DIM + threadIdx.x;
    yIndex = blockIdx.x * TILE_DIM + threadIdx.y;
	
    int index_out = xIndex + (yIndex) * height;
	
    for (int i = 0; i < TILE_DIM; i += BLOCK_ROWS) {
      block[threadIdx.y+i][threadIdx.x] = idata[index_in+i*width];
    }
	
    __syncthreads();
    
	for (int i=0; i<TILE_DIM; i += BLOCK_ROWS) {
		odata[index_out+i*height] = block[threadIdx.y+i][threadIdx.x];
    }
}
  
  












// -----------------------------------------------------------------------------

int main (int argc, char* argv[]){
	
	float kernelTime;
	int size = 1024;
	
	/* for size loop */
	//for (int size = 256; size < 8192; size *= 2){ 
		
	//float *A = (float*) malloc (size * size * sizeof(float));
	//float *B = (float*) malloc (size * size * sizeof(float));
	int *A = (int*) malloc (size * size * sizeof(int));
	int *B = (int*) malloc (size * size * sizeof(int));

	for (int i = 0; i < size*size; i++){
		A[i] = i; //(float) i;
		//B[i] = rand() / 1E6;
	}

	int *d_A, *d_B;

	// CUDA events
    hipEvent_t start, stop;
	
	int size_x, size_y;
	size_x = size_y = size;
  	// execution configuration parameters
    //dim3 grid(size_x/TILE_DIM, size_y/TILE_DIM), threads(TILE_DIM,BLOCK_ROWS);
	dim3 dimGrid(size, 1);
	dim3 dimBlock(1, 1);
	
	if (hipMalloc(&d_A, size*size * sizeof(int)) != hipSuccess){
		printf ("A allocation error !!!\n");
		return 0;
	}
	
	if (hipMalloc(&d_B, size*size * sizeof(int)) != hipSuccess){
		printf ("B allocation error !!!\n");
		hipFree (d_A);
		return 0;
	}
	
	if (hipMemcpy(d_A, A, size*size * sizeof(int), hipMemcpyHostToDevice) != hipSuccess){
		printf ("A copy Error !!!\n");
		hipFree (d_A);
		hipFree (d_B);
		return 0;
	}
	
	if (hipMemcpy(d_B, B, size*size * sizeof(int), hipMemcpyHostToDevice) != hipSuccess){
		printf ("B copy Error !!!\n");
		hipFree (d_A);
		hipFree (d_B);
		return 0;
	}
	


// Part 1 ---------------------------------------------------------------------	
	
	// initialize events, EC parameters
	hipEventCreate (&start);
	hipEventCreate (&stop);
	
	// Warm-up
	//transposeNaive<<<grid, threads>>>(d_B, d_A, size_x, size_y, 'H');
	row_col << <dimGrid, 1 >> > (d_B, d_A, size);
	
	// take measurements for loop over kernel launches
	hipEventRecord(start, 0);
	
	for (int i=0; i < NUM_REPS; i++) {
	  	//transposeNaive<<<grid, threads>>>(d_B, d_A,size_x, size_y, 'H');
		row_col << <dimGrid, 1 >> > (d_B, d_A, size);
	}	
	
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&kernelTime, start, stop);
	
	hipMemcpy(B, d_B, size*size * sizeof(float), hipMemcpyDeviceToHost);
	/*	
	if (hipMemcpy(B, d_B, size*size * sizeof(float), hipMemcpyDeviceToHost) != hipSuccess){
		printf ("Result copy Error !!!\n");
		hipFree (d_A);
		hipFree (d_B);
		return 0;
	}
	*/
	printf ("\nTime for calculation of %d x %d matrix ", size_x, size_y);
	printf ("transpose from rows to columns ");
	printf ("is %f msec.\n", kernelTime/NUM_REPS);


// Part 2 ---------------------------------------------------------------------	
	
	// initialize events, EC parameters
	hipEventCreate (&start);
	hipEventCreate (&stop);
	
	// Warm-up
	//transposeNaive<<<grid, threads>>>(d_B, d_A, size_x, size_y, 'V');
	col_row << <dimGrid, 1 >> > (d_B, d_A, size);
	
	
	// take measurements for loop over kernel launches
	hipEventRecord(start, 0);
	
	for (int i=0; i < NUM_REPS; i++) {
		//transposeNaive<<<grid, threads>>>(d_B, d_A,size_x, size_y, 'V');
		col_row << <dimGrid, 1 >> > (d_B, d_A, size);
	}	
	
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&kernelTime, start, stop);
	
	if (hipMemcpy(B, d_B, size*size * sizeof(float), hipMemcpyDeviceToHost) != hipSuccess){
		printf ("Result copy Error !!!\n");
		hipFree (d_A);
		hipFree (d_B);
		return 0;
	}
	
	hipMemcpy(B, d_B, size*size * sizeof(float), hipMemcpyDeviceToHost);
	
	/*
	printf ("\nTime for calculation of %d x %d matrix ", size_x, size_y);
	printf ("transpose from columns to rows ");
	printf ("is %f msec.\n", kernelTime/NUM_REPS);
	*/
	

// Part 3 ---------------------------------------------------------------------	
	
	// initialize events, EC parameters
	hipEventCreate (&start);
	hipEventCreate (&stop);
	
	// Warm-up
	//transposeNaive<<<grid, threads>>>(d_B, d_A, size_x, size_y, 'H');
	row_col_unroll << <dimGrid, 1 >> > (d_B, d_A, size);	
	
	// take measurements for loop over kernel launches
	hipEventRecord(start, 0);
	
	for (int i=0; i < NUM_REPS; i++) {
	  	//transposeNaive<<<grid, threads>>>(d_B, d_A,size_x, size_y, 'H');
		row_col_unroll << <dimGrid, 1 >> > (d_B, d_A, size);
	}	
	
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&kernelTime, start, stop);
	
	hipMemcpy(B, d_B, size*size * sizeof(float), hipMemcpyDeviceToHost);
	/*	
	if (hipMemcpy(B, d_B, size*size * sizeof(float), hipMemcpyDeviceToHost) != hipSuccess){
		printf ("Result copy Error !!!\n");
		hipFree (d_A);
		hipFree (d_B);
		return 0;
	}
	*/
	printf ("\nTime for calculation of %d x %d matrix ", size_x, size_y);
	printf ("transpose from rows to columns ");
	printf ("is %f msec.\n", kernelTime/NUM_REPS);


// Part 4 ---------------------------------------------------------------------	
	
	// initialize events, EC parameters
	hipEventCreate (&start);
	hipEventCreate (&stop);
	
	// Warm-up
	//transposeNaive<<<grid, threads>>>(d_B, d_A, size_x, size_y, 'V');
	col_row_unroll << <dimGrid, 1 >> > (d_B, d_A, size);
	
	
	// take measurements for loop over kernel launches
	hipEventRecord(start, 0);
	
	for (int i=0; i < NUM_REPS; i++) {
		//transposeNaive<<<grid, threads>>>(d_B, d_A,size_x, size_y, 'V');
		col_row_unroll << <dimGrid, 1 >> > (d_B, d_A, size);
	}	
	
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&kernelTime, start, stop);
	
	if (hipMemcpy(B, d_B, size*size * sizeof(float), hipMemcpyDeviceToHost) != hipSuccess){
		printf ("Result copy Error !!!\n");
		hipFree (d_A);
		hipFree (d_B);
		return 0;
	}
	
	hipMemcpy(B, d_B, size*size * sizeof(float), hipMemcpyDeviceToHost);
	
	
	printf ("\nTime for calculation of %d x %d matrix ", size_x, size_y);
	printf ("transpose from columns to rows ");
	printf ("is %f msec.\n", kernelTime/NUM_REPS);
	
	
		
// Part 5 ---------------------------------------------------------------------	
	
	// initialize events, EC parameters
	hipEventCreate (&start);
	hipEventCreate (&stop);
	
	// Warm-up
	//transposeDiagonal<<<grid, threads>>>(d_B, d_A, size_x, size_y, 'H');
	row_col_diag <<< dimGrid, 1 >>> (d_B, d_A, size);
	
	
	// take measurements for loop over kernel launches
	hipEventRecord(start, 0);
	
	for (int i=0; i < NUM_REPS; i++) {
	  	//transposeDiagonal<<<grid, threads>>>(d_B, d_A,size_x, size_y, 'H');
		row_col_diag <<< dimGrid, 1 >>> (d_B, d_A, size);
	}	
	
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&kernelTime, start, stop);
	
	if (hipMemcpy(B, d_B, size*size * sizeof(float), hipMemcpyDeviceToHost) != hipSuccess){
		printf ("Result copy Error !!!\n");
		hipFree (d_A);
		hipFree (d_B);
		return 0;
	}
	
	printf ("\nTime for calculation of %d x %d matrix ", size_x, size_y);
	printf ("diagonal transpose from rows to columns ");
	printf ("is %f msec.\n", kernelTime/NUM_REPS);
	



// Part 6 ---------------------------------------------------------------------	
	
	// initialize events, EC parameters
	hipEventCreate (&start);
	hipEventCreate (&stop);
	
	// Warm-up
	//transposeDiagonal<<<grid, threads>>>(d_B, d_A, size_x, size_y, 'V');
	col_row_diag << < dimGrid, 1 >> > (d_B, d_A, size);
	
	
	// take measurements for loop over kernel launches
	hipEventRecord(start, 0);
	
	for (int i=0; i < NUM_REPS; i++) {
	  	//transposeDiagonal<<<grid, threads>>>(d_B, d_A,size_x, size_y, 'V');
  		col_row_diag << < dimGrid, 1 >> > (d_B, d_A, size);
	}	
	
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&kernelTime, start, stop);
	
	if (hipMemcpy(B, d_B, size*size * sizeof(float), hipMemcpyDeviceToHost) != hipSuccess){
		printf ("Result copy Error !!!\n");
		hipFree (d_A);
		hipFree (d_B);
		return 0;
	}
	
	printf ("\nTime for calculation of %d x %d matrix ", size_x, size_y);
	printf ("diagonal transpose from columns to rows ");
	printf ("is %f msec.\n", kernelTime/NUM_REPS);
	

	


// Finalize -------------------------------------------------------------------	
	
	printf ("\n\n");
	
	for (int i = 0; i < 5; i++){
		for (int j = 0; j < 5; j++){
			printf ("%4.2f   ", A[i*size+j]);
		}
		printf ("\n");
	}
	
	printf ("\n\n");
	
	for (int i = 0; i < 5; i++){
		for (int j = 0; j < 5; j++){
			printf ("%4.2f\t", B[i*size+j]);
		}
		printf ("\n");
	}
	
	
	free (A);
	free (B);
	hipFree (d_A);
	hipFree (d_B);
	hipEventDestroy(start); 
	hipEventDestroy(stop);
	
	//} /* for size loop */
	
	return 0;
}












